#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h> 
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define SCAN_BLOCK_DIM   256  // needed by sharedMemExclusiveScan implementation
#include "exclusiveScan.cu_inl"


#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS)) 
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif


////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;
    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    float* radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff) {
        radius[index] = 0.02f;
    } else {
        radius[index] += 0.01f;
    }
}


// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() {
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // This conditional is in the inner loop, but it evaluates the
    // same direction for all threads so it's cost is not so
    // bad. Attempting to hoist this conditional is not a required
    // student optimization in Assignment 2

    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;


    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read


    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION

}

/*
// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.  
__global__ void kernelRenderCircles() {
    //int table[1024][1024] = {0};
    __shared__ int table[1024][1024];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //__shared__ float shmImgPtr[256][180];
    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    //printf("screenMaxX - screenMinX: %d\n", screenMaxX- screenMinX);
    // for all pixels in the bonding box

    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
	for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
	    float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
						 invHeight * (static_cast<float>(pixelY) + 0.5f));
	    float diffX = p.x - pixelCenterNorm.x;
	    float diffY = p.y - pixelCenterNorm.y;
	    float pixelDist = diffX * diffX + diffY * diffY;

	    float rad = cuConstRendererParams.radius[index];;
	    float maxDist = rad * rad;

	    // circle does not contribute to the image
	    if (pixelDist <= maxDist)
		table[pixelX][pixelY]++;
	    //shadePixel(index, pixelCenterNorm, p, imgPtr,);//&shmImgPtr[threadIdx.x][4 * a]);
	    //imgPtr++;
	}
    }
}
*/
////////////////////////////////////////////////////////////////////////////////////////
__device__ void prescan(uint *g_odata, uint *g_idata, int n)
{
	__shared__ uint temp[512];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;

	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(ai);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi]; 



	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
 		__syncthreads();
		if (thid < d)
		{ 
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	} 

	if (thid==0) {
		//temp[n – 1 /*+ CONFLICT_FREE_OFFSET(n - 1)*/ ] = 0;
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{ 
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi); 


			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
 	__syncthreads(); 


	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB]; 
} 





__global__ void kernelRenderCircles() {
    /* this queue is intended for remembering circle index */
    int queue[50];
    int queueIndex = 0;

    /* These sharemd memory array will be used in prefixSum function */
    __shared__ uint shmQueue[256];	//input of prefixSum : the role of this array is to divide index of order[] array
    __shared__ uint prefixSum[256];	//output of prefixSum
    __shared__ uint prefixSumScratch[2 * 256];	//The comments inside a prefixSum library file says we need this to calculate it


    /* This array contains circle indices that is colored inside a threa block boundary(32 x 32 pixels),
       and they are sorted by ascending order */
    __shared__ int order[3000];


    /* Statement shown in line 542(extern keyword) used for dynamic allocation of shared memory.
       Reducing the size of shared memory array has positive impact on the execution time.
       From the fact that each image(e.g., rgb, littlebig, rand10k, ...) needs different array size,
       I tried to allocate different array size according to image(e.g., rgb, littlebing, ...),
       but when I use it, it gives me wrong result. I don't know why. */

    //extern __shared__ int order[];

    int blockThreadIndex = blockDim.x * threadIdx.y + threadIdx.x; 

    int numCircles = cuConstRendererParams.numCircles;
    int threadsPerBlock = blockDim.x * blockDim.y;

    /* each thread will handle the number of circles stored in variable 'circle' */ 
    int circle = (numCircles + threadsPerBlock - 1) / threadsPerBlock;


    /* imageX and imageY are the location of image pixels assigned for this thread within boundary. */
    //int imageX = blockIdx.x * blockDim.x + threadIdx.x; // This is intended for assiging each thread 1x1 pixel.
    //int imageY = blockIdx.y * blockDim.y + threadIdx.y; 

    /*Each thread will deal with 2x2 pixels, not 1x1 pixel by multiplying 2.*/
    int imageX = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int imageY = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight; 


    /* Thess variables describe pixel boundary of thread block. */
    
    //int pixelXFrom = blockDim.x * blockIdx.x;	//e.g., 0, 16, 32, ...
    //int pixelXTo = blockDim.x * (blockIdx.x + 1) - 1;	// 15, 31, 63, ...
    //int pixelYFrom = blockDim.y * blockIdx.y;
    //int pixelYTo = blockDim.y * (blockIdx.y + 1) - 1;

    /* Number 2 is intended for 32 x 32 pixels, not 16 x 16 pixels. */
    int pixelXFrom = blockDim.x * blockIdx.x * 2;	//e.g., 0, 64, 128, ...
    int pixelXTo = 2 * blockDim.x * (blockIdx.x + 1) - 1;	// 63, 127, 255, ...
    int pixelYFrom = blockDim.y * blockIdx.y * 2;
    int pixelYTo = 2 * blockDim.y * (blockIdx.y + 1) - 1;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    /* each thread only handles their pixel boundary(2 x 2 pixels),
       and these are used to copy global memory data into local memory. */
    float4 *imgPtr0 = (float4*)(&cuConstRendererParams.imageData[4 * (imageY * imageWidth + imageX)]);
    float4 *imgPtr1 = (float4*)(&cuConstRendererParams.imageData[4 * (imageY * imageWidth + imageX + 1)]);
    float4 *imgPtr2 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX)]);
    float4 *imgPtr3 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX + 1)]);
/*
    float4 *imgPtr4 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX)]);
    float4 *imgPtr5 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX + 1)]);
    float4 *imgPtr6 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX + 2)]);
    float4 *imgPtr7 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 1) * imageWidth + imageX + 3)]);
    float4 *imgPtr8 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 2)* imageWidth + imageX)]);
    float4 *imgPtr9 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 2) * imageWidth + imageX + 1)]);
    float4 *imgPtr10 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 2) * imageWidth + imageX + 2)]);
    float4 *imgPtr11 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 2)* imageWidth + imageX + 3)]);
    float4 *imgPtr12 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 3) * imageWidth + imageX)]);
    float4 *imgPtr13 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 3) * imageWidth + imageX + 1)]);
    float4 *imgPtr14 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 3)* imageWidth + imageX + 2)]);
    float4 *imgPtr15 = (float4*)(&cuConstRendererParams.imageData[4 * ((imageY + 3)* imageWidth + imageX + 3)]);
*/
    /* Copy rgb data in global memory into local memory */
    float4 localImgData0 = *imgPtr0;
    float4 localImgData1 = *imgPtr1;
    float4 localImgData2 = *imgPtr2;
    float4 localImgData3 = *imgPtr3;
/*
    float4 localImgData4 = *imgPtr4;
    float4 localImgData5 = *imgPtr5;
    float4 localImgData6 = *imgPtr6;
    float4 localImgData7 = *imgPtr7;
    float4 localImgData8 = *imgPtr8;
    float4 localImgData9 = *imgPtr9;
    float4 localImgData10 = *imgPtr10;
    float4 localImgData11 = *imgPtr11;
    float4 localImgData12 = *imgPtr12;
    float4 localImgData13 = *imgPtr13;
    float4 localImgData14 = *imgPtr14;
    float4 localImgData15 = *imgPtr15;

*/
    /* Each thread deals with circle indices(From and To) shown in below to
       check whether they are within or across the boundary of this thread block */
    /* When there exist only three circles to be drawn, then each thread has variable
       circleIndexFrom: 0, 1, 2, 3, ... , circleIndexTo: 0, 1, 2, 3, ... , which means
       , in this case, thread number from 3 to 255 will execute for loop described in below.
       However, it doesn't matter because variable "p" and "rad"(in for looop) will have zero valuee */

    int circleIndexFrom = blockThreadIndex * circle;
    int circleIndexTo = (blockThreadIndex + 1) * circle - 1;

    for (int i = circleIndexFrom; i <= circleIndexTo; i++) {
	int index3 = 3 * i;
	float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
	float rad = cuConstRendererParams.radius[i];
	//float newRadWidth = rad * imageWidth;
	//float newRadHeight = rad * imageHeight;	

	/* "rad" is normalized to 0 ~ 1023.xxxxxx */
	float extendXLeft = pixelXFrom - (rad * imageWidth);
	float extendXRight = pixelXTo + (rad * imageWidth);
	float extendYTop = pixelYFrom - (rad * imageHeight);
	float extendYBottom = pixelYTo + (rad * imageHeight);
        /* "circle coordinate" is normailzed to 0 ~ 1023.xxxxxx */
	float circleX = p.x * imageWidth;
	float circleY = p.y * imageHeight;

	
	/* This will check whether the circle index "i" exist within or across the boundary of this thread block's pixels */
	/* Multiplying the value 1.01 and 0.99 is very important to work correctly,
	   Due to the small error from above(maybe the gap between normalized value(~1023) and floating value(0.xxx),
	   I have to multply these constant, it is similar to extend the boundary of thread block's pixel */
	/* I found this fact unexpectedly, because some of the results show me "correctness failed", others "correctness pass" */
	if (extendXLeft <= circleX * 1.01  && extendXRight >= circleX * 0.99 && extendYTop <= circleY * 1.01 && extendYBottom >= circleY * 0.99) {
		queue[queueIndex++] = i;
	}
    }

    shmQueue[blockThreadIndex] = queueIndex;
    __syncthreads();

    /* "prescan" is prefixSum algorithm providied by nVidia. I tried to use this to get
       fast execution time, but failed to get correct result. Maybe I missed something. */
    //prescan(prefixSum, shmQueue, 256);
    //__syncthreads();
    
    /* All threads, together,  in this thread block will calculate prefixSum. */
    sharedMemExclusiveScan(blockThreadIndex, shmQueue, prefixSum, prefixSumScratch, 256);
    __syncthreads();

    /* We have to guarantee that all threads must be located at this point. This is because
       if some of threads are still in shareMemExclusiveScan, which means
       they are still calculating prefixSum, other threads that is executing below code will
       get incorrect value of prefixSum[255] */

    int globalIndex = prefixSum[255] + shmQueue[255];

    int start = prefixSum[blockThreadIndex];
    int end = start + shmQueue[blockThreadIndex];

    //int start = (blockThreadIndex == 0) ? 0 : prefixSum[blockThreadIndex - 1];
    //int end =prefixSum[blockThreadIndex];
    

    int localIndex = 0;

    for (int i = start; i < end; i++) {
	order[i] = queue[localIndex++];
    }
    __syncthreads();
   

    /* Loop circle indices that are stored in shared memory array "order[]" */
    for (int i= 0 ; i < globalIndex; i++) {
	int a = order[i];
	int index3 = 3 * a;
	float3 p = *(float3*)(&cuConstRendererParams.position[index3]);

        /* calculate center point of each pixel which is manged by a thread */
	float2 pixelCenterNorm0 = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
			invHeight * (static_cast<float>(imageY) + 0.5f));
	float2 pixelCenterNorm1 = make_float2(invWidth * (static_cast<float>(imageX + 1) + 0.5f),
			invHeight * (static_cast<float>(imageY) + 0.5f));
	float2 pixelCenterNorm2 = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
			invHeight * (static_cast<float>(imageY+ 1) + 0.5f));
	float2 pixelCenterNorm3 = make_float2(invWidth * (static_cast<float>(imageX + 1) + 0.5f),
			invHeight * (static_cast<float>(imageY + 1) + 0.5f));
/*
	float2 pixelCenterNorm4 = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
			invHeight * (static_cast<float>(imageY + 1) + 0.5f));
	float2 pixelCenterNorm5 = make_float2(invWidth * (static_cast<float>(imageX + 1) + 0.5f),
			invHeight * (static_cast<float>(imageY + 1) + 0.5f));
	float2 pixelCenterNorm6 = make_float2(invWidth * (static_cast<float>(imageX + 2) + 0.5f),
			invHeight * (static_cast<float>(imageY + 1) + 0.5f));
	float2 pixelCenterNorm7 = make_float2(invWidth * (static_cast<float>(imageX + 3) + 0.5f),
			invHeight * (static_cast<float>(imageY + 1) + 0.5f));
	float2 pixelCenterNorm8 = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
			invHeight * (static_cast<float>(imageY + 2) + 0.5f));
	float2 pixelCenterNorm9 = make_float2(invWidth * (static_cast<float>(imageX + 1) + 0.5f),
			invHeight * (static_cast<float>(imageY + 2) + 0.5f));
	float2 pixelCenterNorm10 = make_float2(invWidth * (static_cast<float>(imageX + 2) + 0.5f),
			invHeight * (static_cast<float>(imageY + 2) + 0.5f));
	float2 pixelCenterNorm11 = make_float2(invWidth * (static_cast<float>(imageX + 3) + 0.5f),
			invHeight * (static_cast<float>(imageY + 2) + 0.5f));
	float2 pixelCenterNorm12 = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
			invHeight * (static_cast<float>(imageY + 3) + 0.5f));
	float2 pixelCenterNorm13 = make_float2(invWidth * (static_cast<float>(imageX + 1) + 0.5f),
			invHeight * (static_cast<float>(imageY + 3) + 0.5f));
	float2 pixelCenterNorm14 = make_float2(invWidth * (static_cast<float>(imageX + 2) + 0.5f),
			invHeight * (static_cast<float>(imageY + 3) + 0.5f));
	float2 pixelCenterNorm15 = make_float2(invWidth * (static_cast<float>(imageX + 3) + 0.5f),
			invHeight * (static_cast<float>(imageY + 3) + 0.5f));
*/
	/* each pixel will color RGB in parallel, because each thread has their own range of boundary of pixels */
	shadePixel(a, pixelCenterNorm0, p, &localImgData0);
	shadePixel(a, pixelCenterNorm1, p, &localImgData1);
	shadePixel(a, pixelCenterNorm2, p, &localImgData2);
	shadePixel(a, pixelCenterNorm3, p, &localImgData3);
/*
	shadePixel(a, pixelCenterNorm4, p, &localImgData4);
	shadePixel(a, pixelCenterNorm5, p, &localImgData5);
	shadePixel(a, pixelCenterNorm6, p, &localImgData6);
	shadePixel(a, pixelCenterNorm7, p, &localImgData7);
	shadePixel(a, pixelCenterNorm8, p, &localImgData8);
	shadePixel(a, pixelCenterNorm9, p, &localImgData9);
	shadePixel(a, pixelCenterNorm10, p, &localImgData10);
	shadePixel(a, pixelCenterNorm11, p, &localImgData11);
	shadePixel(a, pixelCenterNorm12, p, &localImgData12);
	shadePixel(a, pixelCenterNorm13, p, &localImgData13);
	shadePixel(a, pixelCenterNorm14, p, &localImgData14);
	shadePixel(a, pixelCenterNorm15, p, &localImgData15);
	//shadePixel(a, pixelCenterNorm2, p, &localImgData2);
	//shadePixel(a, pixelCenterNorm3, p, &localImgData3);
	//shadePixel(a, pixelCenterNorm4, p, &localImgData4);
	//shadePixel(a, pixelCenterNorm, p, &shmImgData[threadIdx.y * 16 + threadIdx.x]);
*/
    }

    /* finally 2x2 pixels' imgData is copied into global memory */
    *imgPtr0 = localImgData0;
    *imgPtr1 = localImgData1;
    *imgPtr2 = localImgData2;
    *imgPtr3 = localImgData3;
/*
    *imgPtr4 = localImgData4;
    *imgPtr5 = localImgData5;
    *imgPtr6 = localImgData6;
    *imgPtr7 = localImgData7;
    *imgPtr8 = localImgData8;
    *imgPtr9 = localImgData9;
    *imgPtr10 = localImgData10;
    *imgPtr11 = localImgData11;
    *imgPtr12 = localImgData12;
    *imgPtr13 = localImgData13;
    *imgPtr14 = localImgData14;
    *imgPtr15 = localImgData15;
*/
}


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {

    // 256 threads per block is a healthy number
	/*
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);
*/
/*
    int size = 2000;
    if (sceneName == CIRCLE_RGB || sceneName == CIRCLE_RGBY)
	size = 300;
    else if (sceneName == CIRCLE_TEST_10K) 
	size = 300;
    else if (sceneName == CIRCLE_TEST_100K)
	size = 1900;
    else
	size = 2800;
   

    printf("before kenrel size: %d\n", size);
*/
    dim3 blockDim(16, 16);
    dim3 gridDim(
        (image->width + (blockDim.x * 2) - 1) / (blockDim.x * 2),
        (image->height + (blockDim.y * 2) - 1) / (blockDim.y * 2));
    kernelRenderCircles<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}
